#include "hip/hip_runtime.h"
/* Copyright (C) 2013-2016, The Regents of The University of Michigan.
All rights reserved.
This software was developed in the APRIL Robotics Lab under the
direction of Edwin Olson, ebolson@umich.edu. This software may be
available under alternative licensing terms; contact the address above.
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
The views and conclusions contained in the software and documentation are those
of the authors and should not be interpreted as representing official policies,
either expressed or implied, of the Regents of The University of Michigan.
*/

#include <math.h>

#include "matd_cuda.cuh"
#include "zarray_cuda.cuh"
#include "homography_cuda.cuh"
#include "math_util_cuda.cuh"

// correspondences is a list of float[4]s, consisting of the points x
// and y concatenated. We will compute a homography such that y = Hx
__device__ matd_cuda_t *homography_compute_cuda(zarray_cuda_t *correspondences, int flags)
{
    // compute centroids of both sets of points (yields a better
    // conditioned information matrix)
    double x_cx = 0, x_cy = 0;
    double y_cx = 0, y_cy = 0;

    for (int i = 0; i < zarray_size_cuda(correspondences); i++) {
        float *c;
        zarray_get_volatile_cuda(correspondences, i, &c);

        x_cx += c[0];
        x_cy += c[1];
        y_cx += c[2];
        y_cy += c[3];
    }

    int sz = zarray_size_cuda(correspondences);
    x_cx /= sz;
    x_cy /= sz;
    y_cx /= sz;
    y_cy /= sz;

    // NB We don't normalize scale; it seems implausible that it could
    // possibly make any difference given the dynamic range of IEEE
    // doubles.

    matd_cuda_t *A = matd_create_cuda(9,9);
    for (int i = 0; i < zarray_size_cuda(correspondences); i++) {
        float *c;
        zarray_get_volatile_cuda(correspondences, i, &c);

        // (below world is "x", and image is "y")
        double worldx = c[0] - x_cx;
        double worldy = c[1] - x_cy;
        double imagex = c[2] - y_cx;
        double imagey = c[3] - y_cy;

        double a03 = -worldx;
        double a04 = -worldy;
        double a05 = -1;
        double a06 = worldx*imagey;
        double a07 = worldy*imagey;
        double a08 = imagey;

        MATD_EL_CUDA(A, 3, 3) += a03*a03;
        MATD_EL_CUDA(A, 3, 4) += a03*a04;
        MATD_EL_CUDA(A, 3, 5) += a03*a05;
        MATD_EL_CUDA(A, 3, 6) += a03*a06;
        MATD_EL_CUDA(A, 3, 7) += a03*a07;
        MATD_EL_CUDA(A, 3, 8) += a03*a08;
        MATD_EL_CUDA(A, 4, 4) += a04*a04;
        MATD_EL_CUDA(A, 4, 5) += a04*a05;
        MATD_EL_CUDA(A, 4, 6) += a04*a06;
        MATD_EL_CUDA(A, 4, 7) += a04*a07;
        MATD_EL_CUDA(A, 4, 8) += a04*a08;
        MATD_EL_CUDA(A, 5, 5) += a05*a05;
        MATD_EL_CUDA(A, 5, 6) += a05*a06;
        MATD_EL_CUDA(A, 5, 7) += a05*a07;
        MATD_EL_CUDA(A, 5, 8) += a05*a08;
        MATD_EL_CUDA(A, 6, 6) += a06*a06;
        MATD_EL_CUDA(A, 6, 7) += a06*a07;
        MATD_EL_CUDA(A, 6, 8) += a06*a08;
        MATD_EL_CUDA(A, 7, 7) += a07*a07;
        MATD_EL_CUDA(A, 7, 8) += a07*a08;
        MATD_EL_CUDA(A, 8, 8) += a08*a08;

        double a10 = worldx;
        double a11 = worldy;
        double a12 = 1;
        double a16 = -worldx*imagex;
        double a17 = -worldy*imagex;
        double a18 = -imagex;

        MATD_EL_CUDA(A, 0, 0) += a10*a10;
        MATD_EL_CUDA(A, 0, 1) += a10*a11;
        MATD_EL_CUDA(A, 0, 2) += a10*a12;
        MATD_EL_CUDA(A, 0, 6) += a10*a16;
        MATD_EL_CUDA(A, 0, 7) += a10*a17;
        MATD_EL_CUDA(A, 0, 8) += a10*a18;
        MATD_EL_CUDA(A, 1, 1) += a11*a11;
        MATD_EL_CUDA(A, 1, 2) += a11*a12;
        MATD_EL_CUDA(A, 1, 6) += a11*a16;
        MATD_EL_CUDA(A, 1, 7) += a11*a17;
        MATD_EL_CUDA(A, 1, 8) += a11*a18;
        MATD_EL_CUDA(A, 2, 2) += a12*a12;
        MATD_EL_CUDA(A, 2, 6) += a12*a16;
        MATD_EL_CUDA(A, 2, 7) += a12*a17;
        MATD_EL_CUDA(A, 2, 8) += a12*a18;
        MATD_EL_CUDA(A, 6, 6) += a16*a16;
        MATD_EL_CUDA(A, 6, 7) += a16*a17;
        MATD_EL_CUDA(A, 6, 8) += a16*a18;
        MATD_EL_CUDA(A, 7, 7) += a17*a17;
        MATD_EL_CUDA(A, 7, 8) += a17*a18;
        MATD_EL_CUDA(A, 8, 8) += a18*a18;

        double a20 = -worldx*imagey;
        double a21 = -worldy*imagey;
        double a22 = -imagey;
        double a23 = worldx*imagex;
        double a24 = worldy*imagex;
        double a25 = imagex;

        MATD_EL_CUDA(A, 0, 0) += a20*a20;
        MATD_EL_CUDA(A, 0, 1) += a20*a21;
        MATD_EL_CUDA(A, 0, 2) += a20*a22;
        MATD_EL_CUDA(A, 0, 3) += a20*a23;
        MATD_EL_CUDA(A, 0, 4) += a20*a24;
        MATD_EL_CUDA(A, 0, 5) += a20*a25;
        MATD_EL_CUDA(A, 1, 1) += a21*a21;
        MATD_EL_CUDA(A, 1, 2) += a21*a22;
        MATD_EL_CUDA(A, 1, 3) += a21*a23;
        MATD_EL_CUDA(A, 1, 4) += a21*a24;
        MATD_EL_CUDA(A, 1, 5) += a21*a25;
        MATD_EL_CUDA(A, 2, 2) += a22*a22;
        MATD_EL_CUDA(A, 2, 3) += a22*a23;
        MATD_EL_CUDA(A, 2, 4) += a22*a24;
        MATD_EL_CUDA(A, 2, 5) += a22*a25;
        MATD_EL_CUDA(A, 3, 3) += a23*a23;
        MATD_EL_CUDA(A, 3, 4) += a23*a24;
        MATD_EL_CUDA(A, 3, 5) += a23*a25;
        MATD_EL_CUDA(A, 4, 4) += a24*a24;
        MATD_EL_CUDA(A, 4, 5) += a24*a25;
        MATD_EL_CUDA(A, 5, 5) += a25*a25;
    }

    // make symmetric
    for (int i = 0; i < 9; i++)
        for (int j = i+1; j < 9; j++)
            MATD_EL_CUDA(A, j, i) = MATD_EL_CUDA(A, i, j);

    matd_cuda_t *H = matd_create_cuda(3,3);

    if (flags & HOMOGRAPHY_COMPUTE_FLAG_INVERSE) {
        // compute singular vector by (carefully) inverting the rank-deficient matrix.

        if (1) {
            matd_cuda_t *Ainv = matd_inverse_cuda(A);
            double scale = 0;

            for (int i = 0; i < 9; i++)
                scale += sq_cuda(MATD_EL_CUDA(Ainv, i, 0));
            scale = sqrt(scale);

            for (int i = 0; i < 3; i++)
                for (int j = 0; j < 3; j++)
                    MATD_EL_CUDA(H, i, j) = MATD_EL_CUDA(Ainv, 3*i+j, 0) / scale;

            matd_destroy_cuda(Ainv);
        } else {

            matd_cuda_t *b = matd_create_data_cuda(9, 1, (double[]) { 1, 0, 0, 0, 0, 0, 0, 0, 0 });
            matd_cuda_t *Ainv = NULL;

            if (0) {
                matd_plu_cuda_t *lu = matd_plu_cuda(A);
                Ainv = matd_plu_solve_cuda(lu, b);
                matd_plu_destroy_cuda(lu);
            } else {
                matd_chol_cuda_t *chol = matd_chol_cuda(A);
                Ainv = matd_chol_solve_cuda(chol, b);
                matd_chol_destroy_cuda(chol);
            }

            double scale = 0;

            for (int i = 0; i < 9; i++)
                scale += sq_cuda(MATD_EL_CUDA(Ainv, i, 0));
            scale = sqrt(scale);

            for (int i = 0; i < 3; i++)
                for (int j = 0; j < 3; j++)
                    MATD_EL_CUDA(H, i, j) = MATD_EL_CUDA(Ainv, 3*i+j, 0) / scale;

            matd_destroy_cuda(b);
            matd_destroy_cuda(Ainv);
        }

    } else {
        // compute singular vector using SVD. A bit slower, but more accurate.
        matd_svd_cuda_t svd = matd_svd_flags_cuda(A, MATD_SVD_NO_WARNINGS);

        for (int i = 0; i < 3; i++)
            for (int j = 0; j < 3; j++)
                MATD_EL_CUDA(H, i, j) = MATD_EL_CUDA(svd.U, 3*i+j, 8);

        matd_destroy_cuda(svd.U);
        matd_destroy_cuda(svd.S);
        matd_destroy_cuda(svd.V);

    }

    matd_cuda_t *Tx = matd_identity_cuda(3);
    MATD_EL_CUDA(Tx,0,2) = -x_cx;
    MATD_EL_CUDA(Tx,1,2) = -x_cy;

    matd_cuda_t *Ty = matd_identity_cuda(3);
    MATD_EL_CUDA(Ty,0,2) = y_cx;
    MATD_EL_CUDA(Ty,1,2) = y_cy;

    matd_cuda_t *H2 = matd_op_cuda("M*M*M", Ty, H, Tx);

    matd_destroy_cuda(A);
    matd_destroy_cuda(Tx);
    matd_destroy_cuda(Ty);
    matd_destroy_cuda(H);

    return H2;
}


// assuming that the projection matrix is:
// [ fx 0  cx 0 ]
// [  0 fy cy 0 ]
// [  0  0  1 0 ]
//
// And that the homography is equal to the projection matrix times the
// model matrix, recover the model matrix (which is returned). Note
// that the third column of the model matrix is missing in the
// expression below, reflecting the fact that the homography assumes
// all points are at z=0 (i.e., planar) and that the element of z is
// thus omitted.  (3x1 instead of 4x1).
//
// [ fx 0  cx 0 ] [ R00  R01  TX ]    [ H00 H01 H02 ]
// [  0 fy cy 0 ] [ R10  R11  TY ] =  [ H10 H11 H12 ]
// [  0  0  1 0 ] [ R20  R21  TZ ] =  [ H20 H21 H22 ]
//                [  0    0    1 ]
//
// fx*R00 + cx*R20 = H00   (note, H only known up to scale; some additional adjustments required; see code.)
// fx*R01 + cx*R21 = H01
// fx*TX  + cx*TZ  = H02
// fy*R10 + cy*R20 = H10
// fy*R11 + cy*R21 = H11
// fy*TY  + cy*TZ  = H12
// R20 = H20
// R21 = H21
// TZ  = H22

__device__ matd_cuda_t *homography_to_pose_cuda(const matd_cuda_t *H, double fx, double fy, double cx, double cy)
{
    // Note that every variable that we compute is proportional to the scale factor of H.
    double R20 = MATD_EL_CUDA(H, 2, 0);
    double R21 = MATD_EL_CUDA(H, 2, 1);
    double TZ  = MATD_EL_CUDA(H, 2, 2);
    double R00 = (MATD_EL_CUDA(H, 0, 0) - cx*R20) / fx;
    double R01 = (MATD_EL_CUDA(H, 0, 1) - cx*R21) / fx;
    double TX  = (MATD_EL_CUDA(H, 0, 2) - cx*TZ)  / fx;
    double R10 = (MATD_EL_CUDA(H, 1, 0) - cy*R20) / fy;
    double R11 = (MATD_EL_CUDA(H, 1, 1) - cy*R21) / fy;
    double TY  = (MATD_EL_CUDA(H, 1, 2) - cy*TZ)  / fy;

    // compute the scale by requiring that the rotation columns are unit length
    // (Use geometric average of the two length vectors we have)
    double length1 = sqrtf(R00*R00 + R10*R10 + R20*R20);
    double length2 = sqrtf(R01*R01 + R11*R11 + R21*R21);
    double s = 1.0 / sqrtf(length1 * length2);

    // get sign of S by requiring the tag to be in front the camera;
    // we assume camera looks in the -Z direction.
    if (TZ > 0)
        s *= -1;

    R20 *= s;
    R21 *= s;
    TZ  *= s;
    R00 *= s;
    R01 *= s;
    TX  *= s;
    R10 *= s;
    R11 *= s;
    TY  *= s;

    // now recover [R02 R12 R22] by noting that it is the cross product of the other two columns.
    double R02 = R10*R21 - R20*R11;
    double R12 = R20*R01 - R00*R21;
    double R22 = R00*R11 - R10*R01;

    // Improve rotation matrix by applying polar decomposition.
    if (1) {
        // do polar decomposition. This makes the rotation matrix
        // "proper", but probably increases the reprojection error. An
        // iterative alignment step would be superior.

        matd_cuda_t *R = matd_create_data_cuda(3, 3, (double[]) { R00, R01, R02,
                                                       R10, R11, R12,
                                                       R20, R21, R22 });

        matd_svd_cuda_t svd = matd_svd_cuda(R);
        matd_destroy_cuda(R);

        R = matd_op_cuda("M*M'", svd.U, svd.V);

        matd_destroy_cuda(svd.U);
        matd_destroy_cuda(svd.S);
        matd_destroy_cuda(svd.V);

        R00 = MATD_EL_CUDA(R, 0, 0);
        R01 = MATD_EL_CUDA(R, 0, 1);
        R02 = MATD_EL_CUDA(R, 0, 2);
        R10 = MATD_EL_CUDA(R, 1, 0);
        R11 = MATD_EL_CUDA(R, 1, 1);
        R12 = MATD_EL_CUDA(R, 1, 2);
        R20 = MATD_EL_CUDA(R, 2, 0);
        R21 = MATD_EL_CUDA(R, 2, 1);
        R22 = MATD_EL_CUDA(R, 2, 2);

        matd_destroy_cuda(R);
    }

    return matd_create_data_cuda(4, 4, (double[]) { R00, R01, R02, TX,
                                               R10, R11, R12, TY,
                                               R20, R21, R22, TZ,
                                                0, 0, 0, 1 });
}

// Similar to above
// Recover the model view matrix assuming that the projection matrix is:
//
// [ F  0  A  0 ]     (see glFrustrum)
// [ 0  G  B  0 ]
// [ 0  0  C  D ]
// [ 0  0 -1  0 ]

__device__ matd_cuda_t *homography_to_model_view_cuda(const matd_cuda_t *H, double F, double G, double A, double B)
{
    // Note that every variable that we compute is proportional to the scale factor of H.
    double R20 = -MATD_EL_CUDA(H, 2, 0);
    double R21 = -MATD_EL_CUDA(H, 2, 1);
    double TZ  = -MATD_EL_CUDA(H, 2, 2);
    double R00 = (MATD_EL_CUDA(H, 0, 0) - A*R20) / F;
    double R01 = (MATD_EL_CUDA(H, 0, 1) - A*R21) / F;
    double TX  = (MATD_EL_CUDA(H, 0, 2) - A*TZ)  / F;
    double R10 = (MATD_EL_CUDA(H, 1, 0) - B*R20) / G;
    double R11 = (MATD_EL_CUDA(H, 1, 1) - B*R21) / G;
    double TY  = (MATD_EL_CUDA(H, 1, 2) - B*TZ)  / G;

    // compute the scale by requiring that the rotation columns are unit length
    // (Use geometric average of the two length vectors we have)
    double length1 = sqrtf(R00*R00 + R10*R10 + R20*R20);
    double length2 = sqrtf(R01*R01 + R11*R11 + R21*R21);
    double s = 1.0 / sqrtf(length1 * length2);

    // get sign of S by requiring the tag to be in front of the camera
    // (which is Z < 0) for our conventions.
    if (TZ > 0)
        s *= -1;

    R20 *= s;
    R21 *= s;
    TZ  *= s;
    R00 *= s;
    R01 *= s;
    TX  *= s;
    R10 *= s;
    R11 *= s;
    TY  *= s;

    // now recover [R02 R12 R22] by noting that it is the cross product of the other two columns.
    double R02 = R10*R21 - R20*R11;
    double R12 = R20*R01 - R00*R21;
    double R22 = R00*R11 - R10*R01;

    // TODO XXX: Improve rotation matrix by applying polar decomposition.

    return matd_create_data_cuda(4, 4, (double[]) { R00, R01, R02, TX,
        R10, R11, R12, TY,
        R20, R21, R22, TZ,
        0, 0, 0, 1 });
}

// Only uses the upper 3x3 matrix.
/*
static void matrix_to_quat(const matd_t *R, double q[4])
{
    // see: "from quaternion to matrix and back"

    // trace: get the same result if R is 4x4 or 3x3:
    double T = MATD_EL(R, 0, 0) + MATD_EL(R, 1, 1) + MATD_EL(R, 2, 2) + 1;
    double S = 0;

    double m0  = MATD_EL(R, 0, 0);
    double m1  = MATD_EL(R, 1, 0);
    double m2  = MATD_EL(R, 2, 0);
    double m4  = MATD_EL(R, 0, 1);
    double m5  = MATD_EL(R, 1, 1);
    double m6  = MATD_EL(R, 2, 1);
    double m8  = MATD_EL(R, 0, 2);
    double m9  = MATD_EL(R, 1, 2);
    double m10 = MATD_EL(R, 2, 2);

    if (T > 0.0000001) {
        S = sqrtf(T) * 2;
        q[1] = -( m9 - m6 ) / S;
        q[2] = -( m2 - m8 ) / S;
        q[3] = -( m4 - m1 ) / S;
        q[0] = 0.25 * S;
    } else if ( m0 > m5 && m0 > m10 )  {	// Column 0:
        S  = sqrtf( 1.0 + m0 - m5 - m10 ) * 2;
        q[1] = -0.25 * S;
        q[2] = -(m4 + m1 ) / S;
        q[3] = -(m2 + m8 ) / S;
        q[0] = (m9 - m6 ) / S;
    } else if ( m5 > m10 ) {			// Column 1:
        S  = sqrtf( 1.0 + m5 - m0 - m10 ) * 2;
        q[1] = -(m4 + m1 ) / S;
        q[2] = -0.25 * S;
        q[3] = -(m9 + m6 ) / S;
        q[0] = (m2 - m8 ) / S;
    } else {
        // Column 2:
        S  = sqrtf( 1.0 + m10 - m0 - m5 ) * 2;
        q[1] = -(m2 + m8 ) / S;
        q[2] = -(m9 + m6 ) / S;
        q[3] = -0.25 * S;
        q[0] = (m4 - m1 ) / S;
    }

    double mag2 = 0;
    for (int i = 0; i < 4; i++)
        mag2 += q[i]*q[i];
    double norm = 1.0 / sqrtf(mag2);
    for (int i = 0; i < 4; i++)
        q[i] *= norm;
}
*/

// overwrites upper 3x3 area of matrix M. Doesn't touch any other elements of M.
__device__ void quat_to_matrix_cuda(const double q[4], matd_cuda_t *M)
{
    double w = q[0], x = q[1], y = q[2], z = q[3];

    MATD_EL_CUDA(M, 0, 0) = w*w + x*x - y*y - z*z;
    MATD_EL_CUDA(M, 0, 1) = 2*x*y - 2*w*z;
    MATD_EL_CUDA(M, 0, 2) = 2*x*z + 2*w*y;

    MATD_EL_CUDA(M, 1, 0) = 2*x*y + 2*w*z;
    MATD_EL_CUDA(M, 1, 1) = w*w - x*x + y*y - z*z;
    MATD_EL_CUDA(M, 1, 2) = 2*y*z - 2*w*x;

    MATD_EL_CUDA(M, 2, 0) = 2*x*z - 2*w*y;
    MATD_EL_CUDA(M, 2, 1) = 2*y*z + 2*w*x;
    MATD_EL_CUDA(M, 2, 2) = w*w - x*x - y*y + z*z;
}
