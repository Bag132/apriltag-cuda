#include "hip/hip_runtime.h"
/* Copyright (C) 2013-2016, The Regents of The University of Michigan.
All rights reserved.
This software was developed in the APRIL Robotics Lab under the
direction of Edwin Olson, ebolson@umich.edu. This software may be
available under alternative licensing terms; contact the address above.
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
The views and conclusions contained in the software and documentation are those
of the authors and should not be interpreted as representing official policies,
either expressed or implied, of the Regents of The University of Michigan.
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <stdint.h>

#include "zmaxheap_cuda.cuh"
#include "cuda_helpers.cuh"
// #include "debug_print.h"

#ifdef _WIN32
static inline long int random(void)
{
        return rand();
}
#endif

//                 0
//         1               2
//      3     4        5       6
//     7 8   9 10    11 12   13 14
//
// Children of node i:  2*i+1, 2*i+2
// Parent of node i: (i-1) / 2
//
// Heap property: a parent is greater than (or equal to) its children.

#define MIN_CAPACITY 16

struct zmaxheap_cuda
{
    size_t el_sz;

    int size;
    int alloc;

    float *values;
    char *data;

    void (*swap)(zmaxheap_cuda_t *heap, int a, int b);
};

__device__ static inline void swap_default_cuda(zmaxheap_cuda_t *heap, int a, int b)
{
    float t = heap->values[a];
    heap->values[a] = heap->values[b];
    heap->values[b] = t;

    char *tmp = (char *) malloc(sizeof(char)*heap->el_sz);
    memcpy(tmp, &heap->data[a*heap->el_sz], heap->el_sz);
    memcpy(&heap->data[a*heap->el_sz], &heap->data[b*heap->el_sz], heap->el_sz);
    memcpy(&heap->data[b*heap->el_sz], tmp, heap->el_sz);
    free(tmp);
}

__device__ static inline void swap_pointer_cuda(zmaxheap_cuda_t *heap, int a, int b)
{
    float t = heap->values[a];
    heap->values[a] = heap->values[b];
    heap->values[b] = t;

    void **pp = (void**) heap->data;
    void *tmp = pp[a];
    pp[a] = pp[b];
    pp[b] = tmp;
}


__device__ zmaxheap_cuda_t *zmaxheap_create_cuda(size_t el_sz)
{
    zmaxheap_cuda_t *heap = (zmaxheap_cuda_t *) calloc_cuda(1, sizeof(zmaxheap_cuda_t));
    heap->el_sz = el_sz;

    heap->swap = swap_default_cuda;

    if (el_sz == sizeof(void*))
        heap->swap = swap_pointer_cuda;

    return heap;
}

__device__ void zmaxheap_destroy_cuda(zmaxheap_cuda_t *heap)
{
    free(heap->values);
    free(heap->data);
    memset(heap, 0, sizeof(zmaxheap_cuda_t));
    free(heap);
}

__device__ int zmaxheap_size_cuda(zmaxheap_cuda_t *heap)
{
    return heap->size;
}

__device__ void zmaxheap_ensure_capacity_cuda(zmaxheap_cuda_t *heap, int capacity)
{
    if (heap->alloc >= capacity)
        return;

    int old_len = heap->alloc;

    int newcap = heap->alloc;

    while (newcap < capacity) {
        if (newcap < MIN_CAPACITY) {
            newcap = MIN_CAPACITY;
            continue;
        }

        newcap *= 2;
    }

    heap->values = (float *) realloc_cuda(heap->values, old_len * sizeof(float), newcap * sizeof(float));
    heap->data = (char *) realloc_cuda(heap->data, old_len * heap->el_sz, newcap * heap->el_sz);
    heap->alloc = newcap;
}

__device__ void zmaxheap_add_cuda(zmaxheap_cuda_t *heap, void *p, float v)
{

    assert (isfinite(v) && "zmaxheap_add: Trying to add non-finite number to heap.  NaN's prohibited, could allow INF with testing");
    zmaxheap_ensure_capacity_cuda(heap, heap->size + 1);

    int idx = heap->size;

    heap->values[idx] = v;
    memcpy(&heap->data[idx*heap->el_sz], p, heap->el_sz);

    heap->size++;

    while (idx > 0) {

        int parent = (idx - 1) / 2;

        // we're done!
        if (heap->values[parent] >= v)
            break;

        // else, swap and recurse upwards.
        heap->swap(heap, idx, parent);
        idx = parent;
    }
}

__device__ void zmaxheap_vmap_cuda(zmaxheap_cuda_t *heap, void (*f)(void*))
{
    assert(heap != NULL);
    assert(f != NULL);
    assert(heap->el_sz == sizeof(void*));

    for (int idx = 0; idx < heap->size; idx++) {
        void *p = NULL;
        memcpy(&p, &heap->data[idx*heap->el_sz], heap->el_sz);
        if (p == NULL) {
            printf("Warning: zmaxheap_vmap item %d is NULL\n", idx);
        }
        f(p);
    }
}

// Removes the item in the heap at the given index.  Returns 1 if the
// item existed. 0 Indicates an invalid idx (heap is smaller than
// idx). This is mostly intended to be used by zmaxheap_remove_max.
__device__ int zmaxheap_remove_index_cuda(zmaxheap_cuda_t *heap, int idx, void *p, float *v)
{
    if (idx >= heap->size)
        return 0;

    // copy out the requested element from the heap.
    if (v != NULL)
        *v = heap->values[idx];
    if (p != NULL)
        memcpy(p, &heap->data[idx*heap->el_sz], heap->el_sz);

    heap->size--;

    // If this element is already the last one, then there's nothing
    // for us to do.
    if (idx == heap->size)
        return 1;

    // copy last element to first element. (which probably upsets
    // the heap property).
    heap->values[idx] = heap->values[heap->size];
    memcpy(&heap->data[idx*heap->el_sz], &heap->data[heap->el_sz * heap->size], heap->el_sz);

    // now fix the heap. Note, as we descend, we're "pushing down"
    // the same node the entire time. Thus, while the index of the
    // parent might change, the parent_score doesn't.
    int parent = idx;
    float parent_score = heap->values[idx];

    // descend, fixing the heap.
    while (parent < heap->size) {

        int left = 2*parent + 1;
        int right = left + 1;

//            assert(parent_score == heap->values[parent]);

        float left_score = (left < heap->size) ? heap->values[left] : -INFINITY;
        float right_score = (right < heap->size) ? heap->values[right] : -INFINITY;

        // put the biggest of (parent, left, right) as the parent.

        // already okay?
        if (parent_score >= left_score && parent_score >= right_score)
            break;

        // if we got here, then one of the children is bigger than the parent.
        if (left_score >= right_score) {
            assert(left < heap->size);
            heap->swap(heap, parent, left);
            parent = left;
        } else {
            // right_score can't be less than left_score if right_score is -INFINITY.
            assert(right < heap->size);
            heap->swap(heap, parent, right);
            parent = right;
        }
    }

    return 1;
}

__device__ int zmaxheap_remove_max_cuda(zmaxheap_cuda_t *heap, void *p, float *v)
{
    return zmaxheap_remove_index_cuda(heap, 0, p, v);
}

__device__ void zmaxheap_iterator_init_cuda(zmaxheap_cuda_t *heap, zmaxheap_iterator_cuda_t *it)
{
    memset(it, 0, sizeof(zmaxheap_iterator_cuda_t));
    it->heap = heap;
    it->in = 0;
    it->out = 0;
}

__device__ int zmaxheap_iterator_next_cuda(zmaxheap_iterator_cuda_t *it, void *p, float *v)
{
    zmaxheap_cuda_t *heap = it->heap;

    if (it->in >= zmaxheap_size_cuda(heap))
        return 0;

    *v = heap->values[it->in];
    memcpy(p, &heap->data[it->in*heap->el_sz], heap->el_sz);

    if (it->in != it->out) {
        heap->values[it->out] = heap->values[it->in];
        memcpy(&heap->data[it->out*heap->el_sz], &heap->data[it->in*heap->el_sz], heap->el_sz);
    }

    it->in++;
    it->out++;
    return 1;
}

__device__ int zmaxheap_iterator_next_volatile_cuda(zmaxheap_iterator_cuda_t *it, void *p, float *v)
{
    zmaxheap_cuda_t *heap = it->heap;

    if (it->in >= zmaxheap_size_cuda(heap))
        return 0;

    *v = heap->values[it->in];
    *((void**) p) = &heap->data[it->in*heap->el_sz];

    if (it->in != it->out) {
        heap->values[it->out] = heap->values[it->in];
        memcpy(&heap->data[it->out*heap->el_sz], &heap->data[it->in*heap->el_sz], heap->el_sz);
    }

    it->in++;
    it->out++;
    return 1;
}

__device__ void zmaxheap_iterator_remove_cuda(zmaxheap_iterator_cuda_t *it)
{
    it->out--;
}

__device__ static void maxheapify_cuda(zmaxheap_cuda_t *heap, int parent)
{
    int left = 2*parent + 1;
    int right = 2*parent + 2;

    int betterchild = parent;

    if (left < heap->size && heap->values[left] > heap->values[betterchild])
        betterchild = left;
    if (right < heap->size && heap->values[right] > heap->values[betterchild])
        betterchild = right;

    if (betterchild != parent) {
        heap->swap(heap, parent, betterchild);
        maxheapify_cuda(heap, betterchild);
    }
}

#if 0 //won't compile if defined but not used
// test the heap property
static void validate(zmaxheap_t *heap)
{
    for (int parent = 0; parent < heap->size; parent++) {
        int left = 2*parent + 1;
        int right = 2*parent + 2;

        if (left < heap->size) {
            assert(heap->values[parent] > heap->values[left]);
        }

        if (right < heap->size) {
            assert(heap->values[parent] > heap->values[right]);
        }
    }
}
#endif
__device__ void zmaxheap_iterator_finish_cuda(zmaxheap_iterator_cuda_t *it)
{
    // if nothing was removed, no work to do.
    if (it->in == it->out)
        return;

    zmaxheap_cuda_t *heap = it->heap;

    heap->size = it->out;

    // restore heap property
    for (int i = heap->size/2 - 1; i >= 0; i--)
        maxheapify_cuda(heap, i);
}
