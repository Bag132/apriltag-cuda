#include "hip/hip_runtime.h"
/* Copyright (C) 2013-2016, The Regents of The University of Michigan.
All rights reserved.
This software was developed in the APRIL Robotics Lab under the
direction of Edwin Olson, ebolson@umich.edu. This software may be
available under alternative licensing terms; contact the address above.
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
The views and conclusions contained in the software and documentation are those
of the authors and should not be interpreted as representing official policies,
either expressed or implied, of the Regents of The University of Michigan.
*/

// limitation: image size must be <32768 in width and height. This is
// because we use a fixed-point 16 bit integer representation with one
// fractional bit.
#define _USE_MATH_DEFINES
#include <math.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

// #include "zmaxheap.h"
// #include "debug_print.h"

#include "apriltag_cuda.cuh"
#include "common/cuda/image_u8x3_cuda.cuh"
#include "common/cuda/zarray_cuda.cuh"
#include "common/cuda/unionfind_cuda.cuh"
// #include "common/timeprofile.h"
#include "common/cuda/zmaxheap_cuda.cuh"
// #include "common/postscript_utils.h"
#include "common/cuda/math_util_cuda.cuh"
#include <hiprand/hiprand_kernel.h>

#include "common/cuda/cuda_helpers.cuh"

#define MAX(a, b) (a > b ? a : b)
#define MIN(a, b) (a < b ? a : b)

///////////////////////////////////////////////////////////////////////////////////////////////////////
//
// START apriltag.h CUDA version
//
///////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ inline uint32_t u64hash_2_cuda(uint64_t x) {
    return (2654435761 * x) >> 32;
}

struct uint64_zarray_cuda_entry
{
    uint64_t id;
    zarray_cuda_t *cluster;

    struct uint64_zarray_cuda_entry *next;
};

struct pt
{
    // Note: these represent 2*actual value.
    uint16_t x, y;
    int16_t gx, gy;

    float slope;
};

struct remove_vertex
{
    int i;           // which vertex to remove?
    int left, right; // left vertex, right vertex

    double err;
};

struct segment
{
    int is_vertex;

    // always greater than zero, but right can be > size, which denotes
    // a wrap around back to the beginning of the points. and left < right.
    int left, right;
};

struct line_fit_pt
{
    double Mx, My;
    double Mxx, Myy, Mxy;
    double W; // total weight
};

struct cluster_hash
{
    uint32_t hash;
    uint64_t id;
    zarray_cuda_t* data;
};

///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////
//
// START CUDA version
//
///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ float pt_compare_angle(struct pt *a, struct pt *b) {
    return a->slope - b->slope;
}

__device__ int err_compare_descending(const void *_a, const void *_b)
{
    const double *a = (const double*) _a;
    const double *b = (const double*) _b;

    return ((*a) < (*b)) ? 1 : -1;
}

// lfps contains *cumulative* moments for N points, with
// index j reflecting points [0,j] (inclusive).
//
// fit a line to the points [i0, i1] (inclusive). i0, i1 are both [0,
// sz) if i1 < i0, we treat this as a wrap around.
__device__ void fit_line(struct line_fit_pt *lfps, int sz, int i0, int i1, double *lineparm, double *err, double *mse)
{
    assert(i0 != i1);
    assert(i0 >= 0 && i1 >= 0 && i0 < sz && i1 < sz);

    double Mx, My, Mxx, Myy, Mxy, W;
    int N; // how many points are included in the set?

    if (i0 < i1) {
        N = i1 - i0 + 1;

        Mx  = lfps[i1].Mx;
        My  = lfps[i1].My;
        Mxx = lfps[i1].Mxx;
        Mxy = lfps[i1].Mxy;
        Myy = lfps[i1].Myy;
        W   = lfps[i1].W;

        if (i0 > 0) {
            Mx  -= lfps[i0-1].Mx;
            My  -= lfps[i0-1].My;
            Mxx -= lfps[i0-1].Mxx;
            Mxy -= lfps[i0-1].Mxy;
            Myy -= lfps[i0-1].Myy;
            W   -= lfps[i0-1].W;
        }

    } else {
        // i0 > i1, e.g. [15, 2]. Wrap around.
        assert(i0 > 0);

        Mx  = lfps[sz-1].Mx   - lfps[i0-1].Mx;
        My  = lfps[sz-1].My   - lfps[i0-1].My;
        Mxx = lfps[sz-1].Mxx  - lfps[i0-1].Mxx;
        Mxy = lfps[sz-1].Mxy  - lfps[i0-1].Mxy;
        Myy = lfps[sz-1].Myy  - lfps[i0-1].Myy;
        W   = lfps[sz-1].W    - lfps[i0-1].W;

        Mx  += lfps[i1].Mx;
        My  += lfps[i1].My;
        Mxx += lfps[i1].Mxx;
        Mxy += lfps[i1].Mxy;
        Myy += lfps[i1].Myy;
        W   += lfps[i1].W;

        N = sz - i0 + i1 + 1;
    }

    assert(N >= 2);

    double Ex = Mx / W;
    double Ey = My / W;
    double Cxx = Mxx / W - Ex*Ex;
    double Cxy = Mxy / W - Ex*Ey;
    double Cyy = Myy / W - Ey*Ey;

    //if (1) {
    //    // on iOS about 5% of total CPU spent in these trig functions.
    //    // 85 ms per frame on 5S, example.pnm
    //    //
    //    // XXX this was using the double-precision atan2. Was there a case where
    //    // we needed that precision? Seems doubtful.
    //    double normal_theta = .5 * atan2f(-2*Cxy, (Cyy - Cxx));
    //    nx_old = cosf(normal_theta);
    //    ny_old = sinf(normal_theta);
    //}

    // Instead of using the above cos/sin method, pose it as an eigenvalue problem.
    double eig_small = 0.5*(Cxx + Cyy - sqrtf((Cxx - Cyy)*(Cxx - Cyy) + 4*Cxy*Cxy));

    if (lineparm) {
        lineparm[0] = Ex;
        lineparm[1] = Ey;

        double eig = 0.5*(Cxx + Cyy + sqrtf((Cxx - Cyy)*(Cxx - Cyy) + 4*Cxy*Cxy));
        double nx1 = Cxx - eig;
        double ny1 = Cxy;
        double M1 = nx1*nx1 + ny1*ny1;
        double nx2 = Cxy;
        double ny2 = Cyy - eig;
        double M2 = nx2*nx2 + ny2*ny2;

        double nx, ny, M;
        if (M1 > M2) {
            nx = nx1;
            ny = ny1;
            M = M1;
        } else {
            nx = nx2;
            ny = ny2;
            M = M2;
        }

        double length = sqrtf(M);
        if (fabs(length) < 1e-12) {
            lineparm[2] = lineparm[3] = 0;
        }
        else {
            lineparm[2] = nx/length;
            lineparm[3] = ny/length;
        }
    }

    // sum of squared errors =
    //
    // SUM_i ((p_x - ux)*nx + (p_y - uy)*ny)^2
    // SUM_i  nx*nx*(p_x - ux)^2 + 2nx*ny(p_x -ux)(p_y-uy) + ny*ny*(p_y-uy)*(p_y-uy)
    //  nx*nx*SUM_i((p_x -ux)^2) + 2nx*ny*SUM_i((p_x-ux)(p_y-uy)) + ny*ny*SUM_i((p_y-uy)^2)
    //
    //  nx*nx*N*Cxx + 2nx*ny*N*Cxy + ny*ny*N*Cyy

    // sum of squared errors
    if (err)
        *err = N*eig_small;

    // mean squared error
    if (mse)
        *mse = eig_small;
}

// returns 0 if the cluster looks bad.
__device__ int quad_segment_agg(zarray_cuda_t *cluster, struct line_fit_pt *lfps, int indices[4])
{
    int sz = zarray_size_cuda(cluster);

    zmaxheap_cuda_t *heap = zmaxheap_create_cuda(sizeof(struct remove_vertex*));

    // We will initially allocate sz rvs. We then have two types of
    // iterations: some iterations that are no-ops in terms of
    // allocations, and those that remove a vertex and allocate two
    // more children.  This will happen at most (sz-4) times.  Thus we
    // need: sz + 2*(sz-4) entries.

    int rvalloc_pos = 0;
    int rvalloc_size = 3*sz;
    struct remove_vertex *rvalloc = (struct remove_vertex *) calloc_cuda(rvalloc_size, sizeof(struct remove_vertex));

    struct segment *segs = (struct segment *) calloc_cuda(sz, sizeof(struct segment));

    // populate with initial entries
    for (int i = 0; i < sz; i++) {
        struct remove_vertex *rv = &rvalloc[rvalloc_pos++];
        rv->i = i;
        if (i == 0) {
            rv->left = sz-1;
            rv->right = 1;
        } else {
            rv->left  = i-1;
            rv->right = (i+1) % sz;
        }

        fit_line(lfps, sz, rv->left, rv->right, NULL, NULL, &rv->err);

        zmaxheap_add_cuda(heap, &rv, -rv->err);

        segs[i].left = rv->left;
        segs[i].right = rv->right;
        segs[i].is_vertex = 1;
    }

    int nvertices = sz;

    while (nvertices > 4) {
        assert(rvalloc_pos < rvalloc_size);

        struct remove_vertex *rv;
        float err;

        int res = zmaxheap_remove_max_cuda(heap, &rv, &err);
        if (!res)
            return 0;
        assert(res);

        // is this remove_vertex valid? (Or has one of the left/right
        // vertices changes since we last looked?)
        if (!segs[rv->i].is_vertex ||
            !segs[rv->left].is_vertex ||
            !segs[rv->right].is_vertex) {
            continue;
        }

        // we now merge.
        assert(segs[rv->i].is_vertex);

        segs[rv->i].is_vertex = 0;
        segs[rv->left].right = rv->right;
        segs[rv->right].left = rv->left;

        // create the join to the left
        if (1) {
            struct remove_vertex *child = &rvalloc[rvalloc_pos++];
            child->i = rv->left;
            child->left = segs[rv->left].left;
            child->right = rv->right;

            fit_line(lfps, sz, child->left, child->right, NULL, NULL, &child->err);

            zmaxheap_add_cuda(heap, &child, -child->err);
        }

        // create the join to the right
        if (1) {
            struct remove_vertex *child = &rvalloc[rvalloc_pos++];
            child->i = rv->right;
            child->left = rv->left;
            child->right = segs[rv->right].right;

            fit_line(lfps, sz, child->left, child->right, NULL, NULL, &child->err);

            zmaxheap_add_cuda(heap, &child, -child->err);
        }

        // we now have one less vertex
        nvertices--;
    }

    free(rvalloc);
    zmaxheap_destroy_cuda(heap);

    int idx = 0;
    for (int i = 0; i < sz; i++) {
        if (segs[i].is_vertex) {
            indices[idx++] = i;
        }
    }

    free(segs);

    return 1;
}


/*

  1. Identify A) white points near a black point and B) black points near a white point.

  2. Find the connected components within each of the classes above,
  yielding clusters of "white-near-black" and
  "black-near-white". (These two classes are kept separate). Each
  segment has a unique id.

  3. For every pair of "white-near-black" and "black-near-white"
  clusters, find the set of points that are in one and adjacent to the
  other. In other words, a "boundary" layer between the two
  clusters. (This is actually performed by iterating over the pixels,
  rather than pairs of clusters.) Critically, this helps keep nearby
  edges from becoming connected.
*/
__device__ int quad_segment_maxima(apriltag_detector_cuda_t *td, zarray_cuda_t *cluster, struct line_fit_pt *lfps, int indices[4])
{
    int sz = zarray_size_cuda(cluster);

    // ksz: when fitting points, how many points on either side do we consider?
    // (actual "kernel" width is 2ksz).
    //
    // This value should be about: 0.5 * (points along shortest edge).
    //
    // If all edges were equally-sized, that would give a value of
    // sz/8. We make it somewhat smaller to account for tags at high
    // aspects.

    // XXX Tunable. Maybe make a multiple of JPEG block size to increase robustness
    // to JPEG compression artifacts?
    int ksz = imin_cuda(20, sz / 12);

    // can't fit a quad if there are too few points.
    if (ksz < 2)
        return 0;

    double *errs = (double *) malloc(sizeof(double)*sz);

    for (int i = 0; i < sz; i++) {
        fit_line(lfps, sz, (i + sz - ksz) % sz, (i + ksz) % sz, NULL, &errs[i], NULL);
    }

    // apply a low-pass filter to errs
    if (1) {
        double *y = (double *) malloc(sizeof(double)*sz);

        // how much filter to apply?

        // XXX Tunable
        double sigma = 1; // was 3

        // cutoff = exp(-j*j/(2*sigma*sigma));
        // log(cutoff) = -j*j / (2*sigma*sigma)
        // log(cutoff)*2*sigma*sigma = -j*j;

        // how big a filter should we use? We make our kernel big
        // enough such that we represent any values larger than
        // 'cutoff'.

        // XXX Tunable (though not super useful to change)
        double cutoff = 0.05;
        int fsz = sqrt(-log(cutoff)*2*sigma*sigma) + 1;
        fsz = 2*fsz + 1;

        // For default values of cutoff = 0.05, sigma = 3,
        // we have fsz = 17.
        float *f = (float *) malloc(sizeof(float)*fsz);

        for (int i = 0; i < fsz; i++) {
            int j = i - fsz / 2;
            f[i] = exp(-j*j/(2*sigma*sigma));
        }

        for (int iy = 0; iy < sz; iy++) {
            double acc = 0;

            for (int i = 0; i < fsz; i++) {
                acc += errs[(iy + i - fsz / 2 + sz) % sz] * f[i];
            }
            y[iy] = acc;
        }

        memcpy(errs, y, sizeof(double)*sz);
        free(y);
        free(f);
    }

    int *maxima = (int *) malloc(sizeof(int)*sz);
    double *maxima_errs = (double *) malloc(sizeof(double)*sz);
    int nmaxima = 0;

    for (int i = 0; i < sz; i++) {
        if (errs[i] > errs[(i+1)%sz] && errs[i] > errs[(i+sz-1)%sz]) {
            maxima[nmaxima] = i;
            maxima_errs[nmaxima] = errs[i];
            nmaxima++;
        }
    }
    free(errs);

    // if we didn't get at least 4 maxima, we can't fit a quad.
    if (nmaxima < 4){
        free(maxima);
        free(maxima_errs);
        return 0;
    }

    // select only the best maxima if we have too many
    int max_nmaxima = td->qtp.max_nmaxima;

    if (nmaxima > max_nmaxima) {
        double *maxima_errs_copy = (double *) malloc(sizeof(double)*nmaxima);
        memcpy(maxima_errs_copy, maxima_errs, sizeof(double)*nmaxima);

        // throw out all but the best handful of maxima. Sorts descending.
        // qsort(maxima_errs_copy, nmaxima, sizeof(double), err_compare_descending);
        quick_sort_descending_cuda(maxima_errs_copy, 0, nmaxima - 1);

        double maxima_thresh = maxima_errs_copy[max_nmaxima];
        int out = 0;
        for (int in = 0; in < nmaxima; in++) {
            if (maxima_errs[in] <= maxima_thresh)
                continue;
            maxima[out++] = maxima[in];
        }
        nmaxima = out;
        free(maxima_errs_copy);
    }
    free(maxima_errs);

    int best_indices[4];
    double best_error = HUGE_VALF;

    double err01, err12, err23, err30;
    double mse01, mse12, mse23, mse30;
    double params01[4], params12[4];

    // disallow quads where the angle is less than a critical value.
    double max_dot = td->qtp.cos_critical_rad; //25*M_PI/180);

    for (int m0 = 0; m0 < nmaxima - 3; m0++) {
        int i0 = maxima[m0];

        for (int m1 = m0+1; m1 < nmaxima - 2; m1++) {
            int i1 = maxima[m1];

            fit_line(lfps, sz, i0, i1, params01, &err01, &mse01);

            if (mse01 > td->qtp.max_line_fit_mse)
                continue;

            for (int m2 = m1+1; m2 < nmaxima - 1; m2++) {
                int i2 = maxima[m2];

                fit_line(lfps, sz, i1, i2, params12, &err12, &mse12);
                if (mse12 > td->qtp.max_line_fit_mse)
                    continue;

                double dot = params01[2]*params12[2] + params01[3]*params12[3];
                if (fabs(dot) > max_dot)
                    continue;

                for (int m3 = m2+1; m3 < nmaxima; m3++) {
                    int i3 = maxima[m3];

                    fit_line(lfps, sz, i2, i3, NULL, &err23, &mse23);
                    if (mse23 > td->qtp.max_line_fit_mse)
                        continue;

                    fit_line(lfps, sz, i3, i0, NULL, &err30, &mse30);
                    if (mse30 > td->qtp.max_line_fit_mse)
                        continue;

                    double err = err01 + err12 + err23 + err30;
                    if (err < best_error) {
                        best_error = err;
                        best_indices[0] = i0;
                        best_indices[1] = i1;
                        best_indices[2] = i2;
                        best_indices[3] = i3;
                    }
                }
            }
        }
    }

    free(maxima);

    if (best_error == HUGE_VALF)
        return 0;

    for (int i = 0; i < 4; i++)
        indices[i] = best_indices[i];

    if (best_error / sz < td->qtp.max_line_fit_mse)
        return 1;
    return 0;
}

/**
 * Compute statistics that allow line fit queries to be
 * efficiently computed for any contiguous range of indices.
 */
__device__ struct line_fit_pt* compute_lfps(int sz, zarray_cuda_t* cluster, image_u8_cuda_t* im) {
    struct line_fit_pt *lfps = (struct line_fit_pt *) calloc_cuda(sz, sizeof(struct line_fit_pt));

    for (int i = 0; i < sz; i++) {
        struct pt *p;
        zarray_get_volatile_cuda(cluster, i, &p);

        if (i > 0) {
            memcpy(&lfps[i], &lfps[i-1], sizeof(struct line_fit_pt));
        }

        {
            // we now undo our fixed-point arithmetic.
            double delta = 0.5; // adjust for pixel center bias
            double x = p->x * .5 + delta;
            double y = p->y * .5 + delta;
            int ix = x, iy = y;
            double W = 1;

            if (ix > 0 && ix+1 < im->width && iy > 0 && iy+1 < im->height) {
                int grad_x = im->buf[iy * im->stride + ix + 1] -
                    im->buf[iy * im->stride + ix - 1];

                int grad_y = im->buf[(iy+1) * im->stride + ix] -
                    im->buf[(iy-1) * im->stride + ix];

                // XXX Tunable. How to shape the gradient magnitude?
                W = sqrtf((float) (grad_x*grad_x) + (float) (grad_y*grad_y)) + 1;
            }

            double fx = x, fy = y;
            lfps[i].Mx  += W * fx;
            lfps[i].My  += W * fy;
            lfps[i].Mxx += W * fx * fx;
            lfps[i].Mxy += W * fx * fy;
            lfps[i].Myy += W * fy * fy;
            lfps[i].W   += W;
        }
    }
    return lfps;
}


__device__ static inline void ptsort(struct pt *pts, int sz)
{
#define MAYBE_SWAP(arr,apos,bpos)                                   \
    if (pt_compare_angle(&(arr[apos]), &(arr[bpos])) > 0) {                        \
        tmp = arr[apos]; arr[apos] = arr[bpos]; arr[bpos] = tmp;    \
    };

    if (sz <= 1)
        return;

    if (sz == 2) {
        struct pt tmp;
        MAYBE_SWAP(pts, 0, 1);
        return;
    }

    // NB: Using less-branch-intensive sorting networks here on the
    // hunch that it's better for performance.
    if (sz == 3) { // 3 element bubble sort is optimal
        struct pt tmp;
        MAYBE_SWAP(pts, 0, 1);
        MAYBE_SWAP(pts, 1, 2);
        MAYBE_SWAP(pts, 0, 1);
        return;
    }

    if (sz == 4) { // 4 element optimal sorting network.
        struct pt tmp;
        MAYBE_SWAP(pts, 0, 1); // sort each half, like a merge sort
        MAYBE_SWAP(pts, 2, 3);
        MAYBE_SWAP(pts, 0, 2); // minimum value is now at 0.
        MAYBE_SWAP(pts, 1, 3); // maximum value is now at end.
        MAYBE_SWAP(pts, 1, 2); // that only leaves the middle two.
        return;
    }
    if (sz == 5) {
        // this 9-step swap is optimal for a sorting network, but two
        // steps slower than a generic sort.
        struct pt tmp;
        MAYBE_SWAP(pts, 0, 1); // sort each half (3+2), like a merge sort
        MAYBE_SWAP(pts, 3, 4);
        MAYBE_SWAP(pts, 1, 2);
        MAYBE_SWAP(pts, 0, 1);
        MAYBE_SWAP(pts, 0, 3); // minimum element now at 0
        MAYBE_SWAP(pts, 2, 4); // maximum element now at end
        MAYBE_SWAP(pts, 1, 2); // now resort the three elements 1-3.
        MAYBE_SWAP(pts, 2, 3);
        MAYBE_SWAP(pts, 1, 2);
        return;
    }

#undef MAYBE_SWAP

    // a merge sort with temp storage.

    struct pt *tmp = (struct pt *) malloc(sizeof(struct pt) * sz);

    memcpy(tmp, pts, sizeof(struct pt) * sz);

    int asz = sz/2;
    int bsz = sz - asz;

    struct pt *as = &tmp[0];
    struct pt *bs = &tmp[asz];

    ptsort(as, asz);
    ptsort(bs, bsz);

    #define MERGE(apos,bpos)                        \
    if (pt_compare_angle(&(as[apos]), &(bs[bpos])) < 0)        \
        pts[outpos++] = as[apos++];             \
    else                                        \
        pts[outpos++] = bs[bpos++];

    int apos = 0, bpos = 0, outpos = 0;
    while (apos + 8 < asz && bpos + 8 < bsz) {
        MERGE(apos,bpos); MERGE(apos,bpos); MERGE(apos,bpos); MERGE(apos,bpos);
        MERGE(apos,bpos); MERGE(apos,bpos); MERGE(apos,bpos); MERGE(apos,bpos);
    }

    while (apos < asz && bpos < bsz) {
        MERGE(apos,bpos);
    }

    if (apos < asz)
        memcpy(&pts[outpos], &as[apos], (asz-apos)*sizeof(struct pt));
    if (bpos < bsz)
        memcpy(&pts[outpos], &bs[bpos], (bsz-bpos)*sizeof(struct pt));

    free(tmp);

#undef MERGE
}

// return 1 if the quad looks okay, 0 if it should be discarded
__device__ int fit_quad(
        apriltag_detector_cuda_t *td,
        image_u8_cuda_t *im,
        zarray_cuda_t *cluster,
        struct quad_cuda *quad,
        int tag_width,
        bool normal_border,
        bool reversed_border) {
    int res = 0;

    int sz = zarray_size_cuda(cluster);
    if (sz < 24) // Synchronize with later check.
        return 0;

    /////////////////////////////////////////////////////////////
    // Step 1. Sort points so they wrap around the center of the
    // quad. We will constrain our quad fit to simply partition this
    // ordered set into 4 groups.

    // compute a bounding box so that we can order the points
    // according to their angle WRT the center.
    struct pt *p1;
    zarray_get_volatile_cuda(cluster, 0, &p1);
    uint16_t xmax = p1->x;
    uint16_t xmin = p1->x;
    uint16_t ymax = p1->y;
    uint16_t ymin = p1->y;
    for (int pidx = 1; pidx < zarray_size_cuda(cluster); pidx++) {
        struct pt *p;
        zarray_get_volatile_cuda(cluster, pidx, &p);

        if (p->x > xmax) {
            xmax = p->x;
        } else if (p->x < xmin) {
            xmin = p->x;
        }

        if (p->y > ymax) {
            ymax = p->y;
        } else if (p->y < ymin) {
            ymin = p->y;
        }
    }

    if ((xmax - xmin)*(ymax - ymin) < tag_width) {
        return 0;
    }

    // add some noise to (cx,cy) so that pixels get a more diverse set
    // of theta estimates. This will help us remove more points.
    // (Only helps a small amount. The actual noise values here don't
    // matter much at all, but we want them [-1, 1]. (XXX with
    // fixed-point, should range be bigger?)
    float cx = (xmin + xmax) * 0.5 + 0.05118;
    float cy = (ymin + ymax) * 0.5 + -0.028581;

    float dot = 0;

    float quadrants[2][2] = {{-1*(2 << 15), 0}, {2*(2 << 15), 2 << 15}};

    for (int pidx = 0; pidx < zarray_size_cuda(cluster); pidx++) {
        struct pt *p;
        zarray_get_volatile_cuda(cluster, pidx, &p);

        float dx = p->x - cx;
        float dy = p->y - cy;

        dot += dx*p->gx + dy*p->gy;

        float quadrant = quadrants[dy > 0][dx > 0];
        if (dy < 0) {
            dy = -dy;
            dx = -dx;
        }

        if (dx < 0) {
            float tmp = dx;
            dx = dy;
            dy = -tmp;
        }
        p->slope = quadrant + dy/dx;
    }

    // Ensure that the black border is inside the white border.
    quad->reversed_border = dot < 0;
    if (!reversed_border && quad->reversed_border) {
        return 0;
    }
    if (!normal_border && !quad->reversed_border) {
        return 0;
    }

    // we now sort the points according to theta. This is a prepatory
    // step for segmenting them into four lines.
    if (1) {
        ptsort((struct pt*) cluster->data, zarray_size_cuda(cluster));
    }

    struct line_fit_pt *lfps = compute_lfps(sz, cluster, im);

    int indices[4];
    if (1) {
        if (!quad_segment_maxima(td, cluster, lfps, indices))
            goto finish;
    } else {
        if (!quad_segment_agg(cluster, lfps, indices))
            goto finish;
    }


    double lines[4][4];

    for (int i = 0; i < 4; i++) {
        int i0 = indices[i];
        int i1 = indices[(i+1)&3];

        double mse;
        fit_line(lfps, sz, i0, i1, lines[i], NULL, &mse);

        if (mse > td->qtp.max_line_fit_mse) {
            res = 0;
            goto finish;
        }
    }

    for (int i = 0; i < 4; i++) {
        // solve for the intersection of lines (i) and (i+1)&3.
        // p0 + lambda0*u0 = p1 + lambda1*u1, where u0 and u1
        // are the line directions.
        //
        // lambda0*u0 - lambda1*u1 = (p1 - p0)
        //
        // rearrange (solve for lambdas)
        //
        // [u0_x   -u1_x ] [lambda0] = [ p1_x - p0_x ]
        // [u0_y   -u1_y ] [lambda1]   [ p1_y - p0_y ]
        //
        // remember that lines[i][0,1] = p, lines[i][2,3] = NORMAL vector.
        // We want the unit vector, so we need the perpendiculars. Thus, below
        // we have swapped the x and y components and flipped the y components.

        double A00 =  lines[i][3],  A01 = -lines[(i+1)&3][3];
        double A10 =  -lines[i][2],  A11 = lines[(i+1)&3][2];
        double B0 = -lines[i][0] + lines[(i+1)&3][0];
        double B1 = -lines[i][1] + lines[(i+1)&3][1];

        double det = A00 * A11 - A10 * A01;

        // inverse.
        if (fabs(det) < 0.001) {
            res = 0;
            goto finish;
        }
        double W00 = A11 / det, W01 = -A01 / det;

        // solve
        double L0 = W00*B0 + W01*B1;

        // compute intersection
        quad->p[i][0] = lines[i][0] + L0*A00;
        quad->p[i][1] = lines[i][1] + L0*A10;

        res = 1;
    }

    // reject quads that are too small
    if (1) {
        double area = 0;

        // get area of triangle formed by points 0, 1, 2, 0
        double length[3], p;
        for (int i = 0; i < 3; i++) {
            int idxa = i; // 0, 1, 2,
            int idxb = (i+1) % 3; // 1, 2, 0
            length[i] = sqrt(sq_cuda(quad->p[idxb][0] - quad->p[idxa][0]) +
                             sq_cuda(quad->p[idxb][1] - quad->p[idxa][1]));
        }
        p = (length[0] + length[1] + length[2]) / 2;

        area += sqrt(p*(p-length[0])*(p-length[1])*(p-length[2]));

        // get area of triangle formed by points 2, 3, 0, 2
        for (int i = 0; i < 3; i++) {
            int idxs[] = { 2, 3, 0, 2 };
            int idxa = idxs[i];
            int idxb = idxs[i+1];
            length[i] = sqrt(sq_cuda(quad->p[idxb][0] - quad->p[idxa][0]) +
                             sq_cuda(quad->p[idxb][1] - quad->p[idxa][1]));
        }
        p = (length[0] + length[1] + length[2]) / 2;

        area += sqrt(p*(p-length[0])*(p-length[1])*(p-length[2]));

        if (area < 0.95*tag_width*tag_width) {
            res = 0;
            goto finish;
        }
    }

    // reject quads whose cumulative angle change isn't equal to 2PI
    if (1) {
        for (int i = 0; i < 4; i++) {
            int i0 = i, i1 = (i+1)&3, i2 = (i+2)&3;

            double dx1 = quad->p[i1][0] - quad->p[i0][0];
            double dy1 = quad->p[i1][1] - quad->p[i0][1];
            double dx2 = quad->p[i2][0] - quad->p[i1][0];
            double dy2 = quad->p[i2][1] - quad->p[i1][1];
            double cos_dtheta = (dx1*dx2 + dy1*dy2)/sqrt((dx1*dx1 + dy1*dy1)*(dx2*dx2 + dy2*dy2));

            if ((cos_dtheta > td->qtp.cos_critical_rad || cos_dtheta < -td->qtp.cos_critical_rad) || dx1*dy2 < dy1*dx2) {
                res = 0;
                goto finish;
            }
        }
    }

  finish:

    free(lfps);

    return res;
}

__device__ static void do_quad_task_cuda(
    apriltag_detector_cuda_t *td, int32_t cidx0, int32_t cidx1, int32_t h, int32_t w, 
    zarray_cuda_t *quads, zarray_cuda_t *clusters, image_u8_cuda_t *im, int32_t tag_width, 
    bool normal_border, bool reversed_border)
{
    for (int cidx = cidx0; cidx < cidx1; cidx++) {

        zarray_cuda_t **cluster;
        zarray_get_volatile_cuda(clusters, cidx, &cluster);

        if (zarray_size_cuda(*cluster) < td->qtp.min_cluster_pixels)
            continue;

        // a cluster should contain only boundary points around the
        // tag. it cannot be bigger than the whole screen. (Reject
        // large connected blobs that will be prohibitively slow to
        // fit quads to.) A typical point along an edge is added two
        // times (because it has 2 unique neighbors). The maximum
        // perimeter is 2w+2h.
        if (zarray_size_cuda(*cluster) > 2*(2*w+2*h)) {
            continue;
        }

        struct quad_cuda quad;
        memset(&quad, 0, sizeof(struct quad_cuda));

        if (fit_quad(td, im, *cluster, &quad, tag_width, normal_border, reversed_border)) {
            // pthread_mutex_lock(&td->mutex); // FIXME
            zarray_add_cuda(quads, &quad);
            // pthread_mutex_unlock(&td->mutex);
        }
    }
}

__device__ zarray_cuda_t* fit_quads(apriltag_detector_cuda_t *td, int w, int h, zarray_cuda_t* clusters, image_u8_cuda_t* im, int32_t num_threads) {
    zarray_cuda_t *quads = zarray_create_cuda(sizeof(struct quad_cuda));

    bool normal_border = false;
    bool reversed_border = false;
    int min_tag_width = 1000000;
    for (int i = 0; i < zarray_size_cuda(td->tag_families); i++) {
        apriltag_family_cuda_t* family;
        zarray_get_cuda(td->tag_families, i, &family);
        if (family->width_at_border < min_tag_width) {
            min_tag_width = family->width_at_border;
        }
        normal_border |= !family->reversed_border;
        reversed_border |= family->reversed_border;
    }
    if (td->quad_decimate > 1)
        min_tag_width /= td->quad_decimate;
    if (min_tag_width < 3) {
        min_tag_width = 3;
    }

    int sz = zarray_size_cuda(clusters);

    int32_t chunksize_;
    if (num_threads >= sz) {
        chunksize_ = 1;
    } else {
        chunksize_ = 1 + sz / num_threads;
    }

    if (threadIdx.x < sz) {
        int32_t c0 = threadIdx.x * chunksize_;
        int32_t c1 = imin_cuda(c0 + chunksize_, sz);
        do_quad_task_cuda(td, c0, c1, h, w, quads, clusters, im, min_tag_width, normal_border, reversed_border);
    }

    return quads;
}


__device__ zarray_cuda_t* merge_clusters_cuda(zarray_cuda_t* c1, zarray_cuda_t* c2) {
    zarray_cuda_t* ret = zarray_create_cuda(sizeof(struct cluster_hash*));
    zarray_ensure_capacity_cuda(ret, zarray_size_cuda(c1) + zarray_size_cuda(c2));

    int i1 = 0;
    int i2 = 0;
    int l1 = zarray_size_cuda(c1);
    int l2 = zarray_size_cuda(c2);

    while (i1 < l1 && i2 < l2) {
        struct cluster_hash** h1;
        struct cluster_hash** h2;
        zarray_get_volatile_cuda(c1, i1, &h1);
        zarray_get_volatile_cuda(c2, i2, &h2);

        if ((*h1)->hash == (*h2)->hash && (*h1)->id == (*h2)->id) {
            zarray_add_range_cuda((*h1)->data, (*h2)->data, 0, zarray_size_cuda((*h2)->data));
            zarray_add_cuda(ret, h1);
            i1++;
            i2++;
            zarray_destroy_cuda((*h2)->data);
            free(*h2);
        } else if ((*h2)->hash < (*h1)->hash || ((*h2)->hash == (*h1)->hash && (*h2)->id < (*h1)->id)) {
            zarray_add_cuda(ret, h2);
            i2++;
        } else {
            zarray_add_cuda(ret, h1);
            i1++;
        }
    }

    zarray_add_range_cuda(ret, c1, i1, l1);
    zarray_add_range_cuda(ret, c2, i2, l2);

    zarray_destroy_cuda(c1);
    zarray_destroy_cuda(c2);

    return ret;
}

__device__ void img_create_alignment(uint32_t width_in, uint32_t height_in, uint32_t alignment_in, uint8_t **buf_out, uint32_t *buflen_out, uint32_t *stride_out) 
{
    int stride = width_in;

    if ((stride % alignment_in) != 0)
        stride += alignment_in - (stride % alignment_in);

    uint8_t *buf = (uint8_t *) calloc_cuda(height_in * stride, sizeof(uint8_t));
    *stride_out = stride;
    *buflen_out = height_in * stride * sizeof(uint8_t);
    *buf_out = buf;
}


#define DO_UNIONFIND2_CUDA(dx, dy) if (im->buf[(y + dy)*s + x + dx] == v) unionfind_connect_cuda(uf, y*w + x, (y + dy)*w + x + dx);

__device__ void do_unionfind_first_line_cuda(unionfind_cuda_t *uf, image_u8_cuda_t *im, int32_t w, int32_t s)
{
    int y = 0;
    uint8_t v;

    for (int x = 1; x < w - 1; x++) {
        v = im->buf[y*s + x];

        if (v == 127)
            continue;

        DO_UNIONFIND2_CUDA(-1, 0);
    }
}

// __device__ void do_unionfind_line2_cuda(unionfind_cuda_t *uf, uint8_t *im, int w, int s, int y)
// {
//     assert(y > 0);

//     uint8_t v_m1_m1;
//     uint8_t v_0_m1 = im[(y - 1)*s];
//     uint8_t v_1_m1 = im[(y - 1)*s + 1];
//     uint8_t v_m1_0;
//     uint8_t v = im[y*s];

//     for (int x = 1; x < w - 1; x++) {
//         v_m1_m1 = v_0_m1;
//         v_0_m1 = v_1_m1;
//         v_1_m1 = im[(y - 1)*s + x + 1];
//         v_m1_0 = v;
//         v = im[y*s + x];

//         if (v == 127)
//             continue;

//         // (dx,dy) pairs for 8 connectivity:
//         // (-1, -1)    (0, -1)    (1, -1)
//         // (-1, 0)    (REFERENCE)

//         // DO_UNIONFIND2(-1, 0);
//         uint32_t dx = -1;
//         uint32_t dy = 0;
//         if (im[(y + dy)*s + x + dx] == v) unionfind_connect_cuda(uf, y*w + x, (y + dy)*w + x + dx);


//         if (x == 1 || !((v_m1_0 == v_m1_m1) && (v_m1_m1 == v_0_m1))) {
//             // DO_UNIONFIND2(0, -1);
//             dx = 0;
//             dy = -1;
//             if (im[(y + dy)*s + x + dx] == v) unionfind_connect_cuda(uf, y*w + x, (y + dy)*w + x + dx);

//         }

//         if (v == 255) {
//             if (x == 1 || !(v_m1_0 == v_m1_m1 || v_0_m1 == v_m1_m1) ) {
//                 // DO_UNIONFIND2(-1, -1);
//                 dx = -1;
//                 dy = -1;
//                 if (im[(y + dy)*s + x + dx] == v) unionfind_connect_cuda(uf, y*w + x, (y + dy)*w + x + dx);

//             }
//             if (!(v_0_m1 == v_1_m1)) {
//                 // DO_UNIONFIND2(1, -1);
//                 dx = 1;
//                 dy = -1;
//                 if (im[(y + dy)*s + x + dx] == v) unionfind_connect_cuda(uf, y*w + x, (y + dy)*w + x + dx);
//             }
//         }
//     }
// }

__device__ static void do_unionfind_line2_cuda(unionfind_cuda_t *uf, image_u8_cuda_t *im, int w, int s, int y)
{
    assert(y > 0);

    uint8_t v_m1_m1;
    uint8_t v_0_m1 = im->buf[(y - 1)*s];
    uint8_t v_1_m1 = im->buf[(y - 1)*s + 1];
    uint8_t v_m1_0;
    uint8_t v = im->buf[y*s];

    for (int x = 1; x < w - 1; x++) {
        v_m1_m1 = v_0_m1;
        v_0_m1 = v_1_m1;
        v_1_m1 = im->buf[(y - 1)*s + x + 1];
        v_m1_0 = v;
        v = im->buf[y*s + x];

        if (v == 127)
            continue;

        // (dx,dy) pairs for 8 connectivity:
        // (-1, -1)    (0, -1)    (1, -1)
        // (-1, 0)    (REFERENCE)
        DO_UNIONFIND2_CUDA(-1, 0);

        if (x == 1 || !((v_m1_0 == v_m1_m1) && (v_m1_m1 == v_0_m1))) {
            DO_UNIONFIND2_CUDA(0, -1);
        }

        if (v == 255) {
            if (x == 1 || !(v_m1_0 == v_m1_m1 || v_0_m1 == v_m1_m1) ) {
                DO_UNIONFIND2_CUDA(-1, -1);
            }
            if (!(v_0_m1 == v_1_m1)) {
                DO_UNIONFIND2_CUDA(1, -1);
            }
        }
    }
}

#undef DO_UNIONFIND2


__device__ void do_unionfind_task2_cuda(unionfind_cuda_t *uf, image_u8_cuda_t *im, int32_t w, int32_t s, int32_t y0, int32_t y1)
{
    for (int y = y0; y < y1; y++) {
        do_unionfind_line2_cuda(uf, im, w, s, y);
    }
}

__device__ uint32_t compute_unionfind_hash_cuda(unionfind_cuda_t *uf)
{
    unsigned long hash = 5381;
    int c;

    for (int i = 0; i < uf->maxid + 1; i++) {
        c = uf->parent[i] + uf->size[i];
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
    }

    return hash;
}

__device__ unionfind_cuda_t *connected_components_cuda(image_u8_cuda_t *threshim, uint32_t w, uint32_t h, uint32_t ts, uint32_t num_threads)
{
    __shared__ unionfind_cuda_t *uf;
    
    if (threadIdx.x == 0) {
        uf = unionfind_create_cuda(w * h);
        do_unionfind_first_line_cuda(uf, threshim, w, ts);
    }

    __syncthreads();

    if (threadIdx.x < h) {
        int32_t row_chunk_size;
        if (num_threads > h) {
            row_chunk_size = 1;
        } else {
            row_chunk_size = 1+ h / num_threads;
        }

        if (threadIdx.x == 0) {
            printf("GPU: Chunk size = %d\n", row_chunk_size);
        }

        int32_t row_start = row_chunk_size * threadIdx.x + 1;
        // if (threadIdx.x != 0) {
        //     ++row_start;
        // }
        int32_t row_end = MIN(row_start + row_chunk_size-1, h);


        __syncthreads();
        
        if (row_start < h) {
            // printf("GPU: thread %d, row_start = %d, row_end = %d\n", threadIdx.x, row_start, row_end);
            do_unionfind_task2_cuda(uf, threshim, w, ts, row_start, row_end);
        } else {
            printf("GPU: Excluded thread %d\n", threadIdx.x);
        }
        
        if (threadIdx.x == 0) {
            uint32_t uf_hash = compute_unionfind_hash_cuda(uf);
            printf("GPU: do_unionfind_task2 hash = 0x%X\n", uf_hash);
        }

        __syncthreads();

        uint32_t threshim_hash = compute_image_hash_cuda(threshim);
        uint32_t it = 0;

        if (threadIdx.x == 0) {
            for (int i = 1; i < num_threads - 1; i++) {
                int32_t row = row_chunk_size * i + 1;
                // printf("GPU: thread %d, row - 1 = %d\n", i, row - 1);
                do_unionfind_line2_cuda(uf, threshim, w, ts, row - 1);
                uint32_t uf_hash = compute_unionfind_hash_cuda(uf);
                // printf("GPU: thread %d row - 1 = %d, uf = 0x%X, uf->maxid = %u, *(uf->size) = %u\n", threadIdx.x, row - 1, uf_hash, uf->maxid, *(uf->size));

            }
        }

        if (threadIdx.x == 0) {
            uint32_t uf_hash = compute_unionfind_hash_cuda(uf);
            printf("GPU: do_unionfind_line2_cuda hash = 0x%X\n", uf_hash);
        }

    }

    __syncthreads();

    return uf;
}

__device__ zarray_cuda_t* do_gradient_clusters_cuda(image_u8_cuda_t* threshim, int ts, int y0, int y1, int w, int nclustermap, unionfind_cuda_t* uf, zarray_cuda_t* clusters) {
    struct uint64_zarray_cuda_entry **clustermap = (struct uint64_zarray_cuda_entry **) calloc_cuda(nclustermap, sizeof(struct uint64_zarray_cuda_entry*));

    int mem_chunk_size = 2048;
    struct uint64_zarray_cuda_entry** mem_pools = (struct uint64_zarray_cuda_entry **) malloc(sizeof(struct uint64_zarray_cuda_entry *)*(1 + 2 * nclustermap / mem_chunk_size)); // SmodeTech: avoid memory corruption when nclustermap < mem_chunk_size
    int mem_pool_idx = 0;
    int mem_pool_loc = 0;
    mem_pools[mem_pool_idx] = (struct uint64_zarray_cuda_entry *) calloc_cuda(mem_chunk_size, sizeof(struct uint64_zarray_cuda_entry));

    for (int y = y0; y < y1; y++) {
        bool connected_last = false;
        for (int x = 1; x < w-1; x++) {

            uint8_t v0 = threshim->buf[y*ts + x];
            if (v0 == 127) {
                connected_last = false;
                continue;
            }

            // XXX don't query this until we know we need it?
            uint64_t rep0 = unionfind_get_representative_cuda(uf, y*w + x);
            if (unionfind_get_set_size_cuda(uf, rep0) < 25) {
                connected_last = false;
                continue;
            }

            bool connected;
#define DO_CONN(dx, dy)                                                  \
            if (1) {                                                    \
                uint8_t v1 = threshim->buf[(y + dy)*ts + x + dx];       \
                                                                        \
                if (v0 + v1 == 255) {                                   \
                    uint64_t rep1 = unionfind_get_representative_cuda(uf, (y + dy)*w + x + dx); \
                    if (unionfind_get_set_size_cuda(uf, rep1) > 24) {        \
                        uint64_t clusterid;                                 \
                        if (rep0 < rep1)                                    \
                            clusterid = (rep1 << 32) + rep0;                \
                        else                                                \
                            clusterid = (rep0 << 32) + rep1;                \
                                                                            \
                        /* XXX lousy hash function */                       \
                        uint32_t clustermap_bucket = u64hash_2_cuda(clusterid) % nclustermap; \
                        struct uint64_zarray_cuda_entry *entry = clustermap[clustermap_bucket]; \
                        while (entry && entry->id != clusterid) {           \
                            entry = entry->next;                            \
                        }                                                   \
                                                                            \
                        if (!entry) {                                       \
                            if (mem_pool_loc == mem_chunk_size) {           \
                                mem_pool_loc = 0;                           \
                                mem_pool_idx++;                             \
                                mem_pools[mem_pool_idx] = (struct uint64_zarray_cuda_entry *) calloc_cuda(mem_chunk_size, sizeof(struct uint64_zarray_cuda_entry)); \
                            }                                               \
                            entry = mem_pools[mem_pool_idx] + mem_pool_loc; \
                            mem_pool_loc++;                                 \
                                                                            \
                            entry->id = clusterid;                          \
                            entry->cluster = zarray_create_cuda(sizeof(struct pt)); \
                            entry->next = clustermap[clustermap_bucket];    \
                            clustermap[clustermap_bucket] = entry;          \
                        }                                                   \
                                                                            \
                        struct pt p = { .x = 2*x + dx, .y = 2*y + dy, .gx = dx*((int) v1-v0), .gy = dy*((int) v1-v0)}; \
                        zarray_add_cuda(entry->cluster, &p);                     \
                        connected = true;                                   \
                    }                                                   \
                }                                                       \
            }

            // do 4 connectivity. NB: Arguments must be [-1, 1] or we'll overflow .gx, .gy
            DO_CONN(1, 0);
            DO_CONN(0, 1);

            // do 8 connectivity
            if (!connected_last) {
                // Checking 1, 1 on the previous x, y, and -1, 1 on the current
                // x, y result in duplicate points in the final list.  Only
                // check the potential duplicate if adding this one won't
                // create a duplicate.
                DO_CONN(-1, 1);
            }
            connected = false;
            DO_CONN(1, 1);
            connected_last = connected;
        }
    }
#undef DO_CONN

    for (int i = 0; i < nclustermap; i++) {
        int start = zarray_size_cuda(clusters);
        for (struct uint64_zarray_cuda_entry *entry = clustermap[i]; entry; entry = entry->next) {
            struct cluster_hash* cluster_hash = (struct cluster_hash *) malloc(sizeof(struct cluster_hash));
            cluster_hash->hash = u64hash_2_cuda(entry->id) % nclustermap;
            cluster_hash->id = entry->id;
            cluster_hash->data = entry->cluster;
            zarray_add_cuda(clusters, &cluster_hash);
        }
        int end = zarray_size_cuda(clusters);

        // Do a quick bubblesort on the secondary key.
        int n = end - start;
        for (int j = 0; j < n - 1; j++) {
            for (int k = 0; k < n - j - 1; k++) {
                struct cluster_hash** hash1;
                struct cluster_hash** hash2;
                zarray_get_volatile_cuda(clusters, start + k, &hash1);
                zarray_get_volatile_cuda(clusters, start + k + 1, &hash2);
                if ((*hash1)->id > (*hash2)->id) {
                    struct cluster_hash tmp = **hash2;
                    **hash2 = **hash1;
                    **hash1 = tmp;
                }
            }
        }
    }
    for (int i = 0; i <= mem_pool_idx; i++) {
        free(mem_pools[i]);
    }
    free(mem_pools);
    free(clustermap);

    return clusters;
}

__device__ uint32_t compute_clusters_hash_cuda(zarray_cuda_t *clusters)
{
    unsigned long hash = 5381;
    int c;

    for (int i = 0; i < zarray_size_cuda(clusters); i++) {
        zarray_cuda_t *cluster;
        zarray_get_cuda(clusters, i, &cluster);

        for (int j = 0; j < zarray_size_cuda(cluster); j++) {
            struct pt *p;
            zarray_get_volatile_cuda(cluster, j, &p);
            c += p->x + p->y + p->gx + p->gy + p->slope;
        }

        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
        c = 0;
    }

    return hash;
}

__device__ zarray_cuda_t* gradient_clusters_cuda(apriltag_detector_cuda_t *td, image_u8_cuda_t* threshim, int w, int h, int ts, unionfind_cuda_t* uf, uint32_t num_threads) {
    __shared__ zarray_cuda_t* clusters;
    int nclustermap = 0.2*w*h;

    int sz = h - 1;

    int32_t chunksize;
    if (num_threads >= sz) {
        chunksize = 1;
    } else {
        chunksize = 1 + sz / num_threads;
    }
    
    __shared__ zarray_cuda_t** clusters_list;
    int32_t cluster_list_len = num_threads > sz ? sz : num_threads;
    if (threadIdx.x == 0) {
        clusters_list = (zarray_cuda_t **) malloc(sizeof(zarray_cuda_t *) * cluster_list_len);
    }

    __syncthreads();
    if (threadIdx.x < sz) {
        int32_t y0 = chunksize * threadIdx.x;
        int32_t y1 = y0 + chunksize;

        ++y0;
        ++y1;
        
        y1 = y1 > sz ? sz : y1;

        __syncthreads();

        int ncmap = nclustermap / (sz / chunksize + 1);

        printf("GPU: Clustering thread %d y0 = %d y1 = %d, ts = %d, w = %d, nclustermap = %d\n", threadIdx.x, y0, y1, ts, w, ncmap);
        zarray_cuda_t *clusters = do_gradient_clusters_cuda(threshim, ts, y0, y1, w, ncmap, uf, zarray_create_cuda(sizeof(struct cluster_hash*)));
        if (threadIdx.x == 0) {
            for (int i = 0; i < zarray_size_cuda(clusters); i++) {
                struct cluster_hash chash;
                zarray_get_cuda(clusters, i, &chash);
                printf("GPU: thread %d clusters[%d] = 0x%X\n", threadIdx.x, i, chash.hash);
            }
        }
        clusters_list[threadIdx.x] = clusters;
    }

    __syncthreads(); 

    if (threadIdx.x == 0) {
        int length = cluster_list_len;
        while (length > 1) {
            int write = 0;
            for (int i = 0; i < length - 1; i += 2) {
                clusters_list[write] = merge_clusters_cuda(clusters_list[i], clusters_list[i + 1]);
                write++;
            }

            if (length % 2) {
                clusters_list[write] = clusters_list[length - 1];
            }

            length = (length >> 1) + length % 2;
        }

        if (threadIdx.x == 0) {
            clusters = zarray_create_cuda(sizeof(zarray_cuda_t*));
            zarray_ensure_capacity_cuda(clusters, zarray_size_cuda(clusters_list[0]));
        }

        for (int i = 0; i < zarray_size_cuda(clusters_list[0]); i++) {
            struct cluster_hash** hash;
            zarray_get_volatile_cuda(clusters_list[0], i, &hash);
            zarray_add_cuda(clusters, &(*hash)->data);
            free(*hash);
        }

        zarray_destroy_cuda(clusters_list[0]);
        free(clusters_list);
    }
    return clusters;
}

__device__ void minmax_task_cuda(image_u8_cuda_t *im, uint8_t *im_max, uint8_t *im_min, int32_t ty_start, int32_t ty_end) 
{
    const int tile_size = 4;

    // Tiled img width 
    int tw = im->width / tile_size;
    for (int32_t ty = ty_start; ty < ty_end; ty++) {
        if (threadIdx.x == 0) {
            // printf("first loop\n");
        }

        for (int tx = 0; tx < tw; tx++) {
            uint8_t max = 0, min = 255;
            // if (threadIdx.x == 0) {
            //     printf("second loop tx = %d tw = %d\n", tx, tw);
            // }

            // Iterate inner y pixels
            for (int dy = 0; dy < tile_size; dy++) { // FIXME: runs forever
                // if (threadIdx.x == 0) {
                //     printf("third loop dy = %d, tile_size = %d\n", dy, tile_size);
                // }

                // Iterate inner x pixels
                for (int dx = 0; dx < tile_size; dx++) {
                    // if (threadIdx.x == 0) {
                    //     printf("fourth loop\n"); //embedde and robotics
                    // }

                    // Get current pixel
                    // printf("Accessing %d with buf size %d\n", (ty*tile_size+dy)*im->stride + tx*tile_size + dx, im->stride * im->height);
                    uint8_t v = im->buf[(ty*tile_size+dy)*im->stride + tx*tile_size + dx];
                    // Find min and max pixel values inside the current tile
                    if (v < min)
                        min = v;
                    if (v > max)
                        max = v;
                }
                // return;
            }
            // Set max and min values 
            im_max[ty*tw+tx] = max;
            im_min[ty*tw+tx] = min;
            // return;
        }
        // return;
    }
}

__device__ void blur_task_cuda(image_u8_cuda_t *im, uint8_t *im_max, uint8_t *im_min, uint8_t *im_max_tmp, uint8_t *im_min_tmp, int32_t ty_start, int32_t ty_end)
{
    const int32_t tile_size = 4;
    int32_t tw  = im->width / tile_size;
    int32_t th = im->height / tile_size;

    for (int32_t ty = ty_start; ty < ty_end; ty++) {
        for (int tx = 0; tx < tw; tx++) {
            uint8_t max = 0, min = 255;

            for (int dy = -1; dy <= 1; dy++) {
                if (ty+dy < 0 || ty+dy >= th)
                    continue;
                for (int dx = -1; dx <= 1; dx++) {
                    if (tx+dx < 0 || tx+dx >= tw)
                        continue;

                    uint8_t m = im_max[(ty+dy)*tw+tx+dx];
                    if (m > max)
                        max = m;
                    m = im_min[(ty+dy)*tw+tx+dx];
                    if (m < min)
                        min = m;
                }
            }

            im_max_tmp[ty*tw + tx] = max;
            im_min_tmp[ty*tw + tx] = min;
        }
    }
}

__device__ void threshold_task_cuda(image_u8_cuda_t *im, image_u8_cuda_t *threshim, uint8_t *im_max, uint8_t *im_min, apriltag_detector_cuda_t *td, int32_t ty_start, int32_t ty_end)
{
    const int32_t tilesz = 4;
    int32_t tw = im->width / tilesz;

    for (int32_t ty = ty_start; ty < ty_end; ty++) {
        for (int tx = 0; tx < tw; tx++) {
            int min = im_min[ty*tw + tx];
            int max = im_max[ty*tw + tx];

            // low contrast region? (no edges)
            if (max - min < td->qtp.min_white_black_diff) {
                for (int dy = 0; dy < tilesz; dy++) {
                    int y = ty*tilesz + dy;

                    for (int dx = 0; dx < tilesz; dx++) {
                        int x = tx*tilesz + dx;

                        threshim->buf[y*im->stride+x] = 127;
                    }
                }
                continue;
            }

            // otherwise, actually threshold this tile.

            // argument for biasing towards dark; specular highlights
            // can be substantially brighter than white tag parts
            uint8_t thresh = min + (max - min) / 2;

            for (int dy = 0; dy < tilesz; dy++) {
                int y = ty*tilesz + dy;

                for (int dx = 0; dx < tilesz; dx++) {
                    int x = tx*tilesz + dx;

                    uint8_t v = im->buf[y*im->stride+x];
                    if (v > thresh)
                        threshim->buf[y*im->stride+x] = 255;
                    else
                        threshim->buf[y*im->stride+x] = 0;
                }
            }
        }
    }
}


// void syncthreads(int32_t num_threads) 

__device__ image_u8_cuda_t *threshold_cuda(apriltag_detector_cuda_t *td, image_u8_cuda_t *im, int32_t num_threads) 
{
    int w = im->width, h = im->height, s = im->stride;
    
    const int tilesz = 4;

    int tw = w / tilesz;
    int th = h / tilesz;

    __shared__ image_u8_cuda_t *threshim;
    __shared__ uint8_t *im_max;
    __shared__ uint8_t *im_min;
    __shared__ uint8_t *im_max_tmp;
    __shared__ uint8_t *im_min_tmp;

    if (threadIdx.x == 0) {
        // printf("GPU: Creating threshim\n");
        threshim = image_u8_create_alignment_cuda(w, h, s); 
        im_max = (uint8_t *) calloc_cuda(tw*th, sizeof(uint8_t));
        im_min = (uint8_t *) calloc_cuda(tw*th, sizeof(uint8_t));
        im_max_tmp = (uint8_t *) calloc_cuda(tw*th, sizeof(uint8_t));
        im_min_tmp = (uint8_t *) calloc_cuda(tw*th, sizeof(uint8_t));
    }

    if (threadIdx.x < th) { // FIXME: Illegal. Can't have __syncthreads inside if
        int32_t row_chunk_size;
        if (num_threads > th) {
            row_chunk_size = 1;
        } else {
            row_chunk_size = 1 + th / num_threads;
        }

        int32_t row_start = row_chunk_size * threadIdx.x;
        int32_t row_end = MIN(row_start + row_chunk_size, th);

        __syncthreads(); // FIXME: Jets

        minmax_task_cuda(im, im_max, im_min, row_start, row_end);

        __syncthreads();

        // if (threadIdx.x == 0) {
        //     uint32_t im_max_hash = compute_buf_hash_cuda(im_max, tw * th * sizeof(uint8_t));
        //     uint32_t im_min_hash = compute_buf_hash_cuda(im_min, tw * th * sizeof(uint8_t));
        //     printf("GPU: minmax im_max: 0x%X, im_min: 0x%X\n", im_max_hash, im_min_hash);
        // }

        blur_task_cuda(im, im_max, im_min, im_max_tmp, im_min_tmp, row_start, row_end);

        __syncthreads();

        if (threadIdx.x == 0) {
            free(im_max);
            free(im_min);
            im_max = im_max_tmp;
            im_min = im_min_tmp;

            uint32_t im_max_hash = compute_buf_hash_cuda(im_max, tw * th * sizeof(uint8_t));
            uint32_t im_min_hash = compute_buf_hash_cuda(im_min, tw * th * sizeof(uint8_t));
            printf("GPU: blur im_max: 0x%X, im_min: 0x%X\n", im_max_hash, im_min_hash);
        }

        __syncthreads();

        threshold_task_cuda(im, threshim, im_max, im_min, td, row_start, row_end);

        __syncthreads();

        // if (threadIdx.x == 0) {
        //     uint32_t im_max_hash = compute_buf_hash_cuda(im_max, tw * th * sizeof(uint8_t));
        //     uint32_t im_min_hash = compute_buf_hash_cuda(im_min, tw * th * sizeof(uint8_t));
        //     uint32_t threshim_hash = compute_image_hash_cuda(threshim); 
        //     printf("GPU: threshold im_max: 0x%X, im_min: 0x%X, threshim: 0x%X\n", im_max_hash, im_min_hash, threshim_hash);
        // }
    } else {
        printf("Skipped thresholding\n");
    }

    // we skipped over the non-full-sized tiles above. Fix those now.
    if (threadIdx.x == 0) {
        for (int y = 0; y < h; y++) {

            // what is the first x coordinate we need to process in this row?

            int x0;

            if (y >= th*tilesz) {
                x0 = 0; // we're at the bottom; do the whole row.
            } else {
                x0 = tw*tilesz; // we only need to do the right most part.
            }

            // compute tile coordinates and clamp.
            int ty = y / tilesz;
            if (ty >= th)
                ty = th - 1;

            for (int x = x0; x < w; x++) {
                int tx = x / tilesz;
                if (tx >= tw)
                    tx = tw - 1;

                int max = im_max[ty*tw + tx];
                int min = im_min[ty*tw + tx];
                int thresh = min + (max - min) / 2;

                uint8_t v = im->buf[y*s+x];
                if (v > thresh)
                    threshim->buf[y*s+x] = 255;
                else
                    threshim->buf[y*s+x] = 0;
            }
        }
        
        free(im_min);
        free(im_max);


        // this is a dilate/erode deglitching scheme that does not improve
        // anything as far as I can tell.
        if (td->qtp.deglitch) {
            image_u8_cuda_t *tmp = image_u8_create_cuda(w, h);

            for (int y = 1; y + 1 < h; y++) {
                for (int x = 1; x + 1 < w; x++) {
                    uint8_t max = 0;
                    for (int dy = -1; dy <= 1; dy++) {
                        for (int dx = -1; dx <= 1; dx++) {
                            uint8_t v = threshim->buf[(y+dy)*s + x + dx];
                            if (v > max)
                                max = v;
                        }
                    }
                    tmp->buf[y*s+x] = max;
                }
            }

            for (int y = 1; y + 1 < h; y++) {
                for (int x = 1; x + 1 < w; x++) {
                    uint8_t min = 255;
                    for (int dy = -1; dy <= 1; dy++) {
                        for (int dx = -1; dx <= 1; dx++) {
                            uint8_t v = tmp->buf[(y+dy)*s + x + dx];
                            if (v < min)
                                min = v;
                        }
                    }
                    threshim->buf[y*s+x] = min;
                }
            }

            image_u8_destroy_cuda(tmp);
        }
    }

    __syncthreads();

    // if (threadIdx.x == 0) {
    //     uint32_t threshim_hash = compute_image_hash_cuda(threshim); 
    //     printf("GPU: returning threshim: 0x%X\n", threshim_hash);
    // }

    return threshim;
}

__device__ zarray_cuda_t *apriltag_quad_thresh_cuda(apriltag_detector_cuda_t *td, image_u8_cuda_t *im, int32_t num_threads, image_u8x3_cuda_t **dbg)
{
    ////////////////////////////////////////////////////////
    // step 1. threshold the image, creating the edge image.

    int w = im->width, h = im->height;

    __shared__ image_u8_cuda_t *threshim;

    if (threadIdx.x == 0) {
        threshim = threshold_cuda(td, im, num_threads);
    } else {
        threshold_cuda(td, im, num_threads);
    }

    // *dbg = threshim;


    int ts = threshim->stride;

    ////////////////////////////////////////////////////////
    // step 2. find connected components.

    __shared__ unionfind_cuda_t *uf;
    __syncthreads();
    if (threadIdx.x == 0) {
        uf = connected_components_cuda(threshim, w, h, ts, num_threads);
    } else {
        connected_components_cuda(threshim, w, h, ts, num_threads);
    }

    __syncthreads();


    if (threadIdx.x == 0) {

        if (0) {
            hiprandState_t state;
            hiprand_init(1234, threadIdx.x, 0, &state);
            
            image_u8x3_cuda_t *d = image_u8x3_create_cuda(w, h);

            uint32_t *colors = (uint32_t*) calloc_cuda(w*h, sizeof(*colors));

            for (int y = 0; y < h; y++) {
                for (int x = 0; x < w; x++) {
                    uint32_t v = unionfind_get_representative_cuda(uf, y*w+x);

                    if ((int)unionfind_get_set_size_cuda(uf, v) < td->qtp.min_cluster_pixels)
                        continue;

                    uint32_t color = colors[v];
                    uint8_t r = color >> 16,
                        g = color >> 8,
                        b = color;

                    if (color == 0) {
                        const int bias = 50;
                        uint8_t rand1 = (uint8_t) (hiprand_uniform(&state) * (200-bias));
                        uint8_t rand2 = (uint8_t) (hiprand_uniform(&state) * (200-bias));
                        uint8_t rand3 = (uint8_t) (hiprand_uniform(&state) * (200-bias));
                        // uint8_t rand1 = (uint8_t) (10 * (200-bias));
                        // uint8_t rand2 = (uint8_t) (20 * (200-bias));
                        // uint8_t rand3 = (uint8_t) (30 * (200-bias));

                        // printf("GPU: rand1 = %u, rand2 = %u, rand3 = %u\n", rand1, rand2, rand3);

                        r = bias + rand1;
                        g = bias + rand2;
                        b = bias + rand3;
                        colors[v] = (r << 16) | (g << 8) | b;
                    }

                    d->buf[y*d->stride + 3*x + 0] = r;
                    d->buf[y*d->stride + 3*x + 1] = g;
                    d->buf[y*d->stride + 3*x + 2] = b;
                }
            }

            free(colors);

            *dbg = d;

            uint32_t dbg_hash = compute_image8x3_hash_cuda(*dbg);
            printf("GPU: thread 0 Connected components debug image: 0x%X, w = %d, s = %d, h = %d\n", 
                dbg_hash, (*dbg)->width, (*dbg)->stride, (*dbg)->height);
        }
    }
    
    if (threadIdx.x == 0 || threadIdx.x == 2) {
        uint32_t uf_hash = compute_unionfind_hash_cuda(uf);
        printf("GPU: thread %d Connected components ret = 0x%X\n", threadIdx.x, uf_hash);
    }
    __syncthreads();

    __shared__ zarray_cuda_t* clusters;
    __syncthreads();
    if (threadIdx.x == 0) {
        clusters = gradient_clusters_cuda(td, threshim, w, h, ts, uf, num_threads);
    } else {
        gradient_clusters_cuda(td, threshim, w, h, ts, uf, num_threads);
    }

    __syncthreads();

    if (threadIdx.x == 0 || threadIdx.x == 2) {
        uint32_t chash = compute_clusters_hash_cuda(clusters);
        printf("GPU: thread %d clusters = 0x%X, gradient_cluster hash = 0x%X\n", threadIdx.x, clusters, chash);
    }

    if (threadIdx.x == 0) {
        if (1) {
            hiprandState_t state;
            hiprand_init(1234, threadIdx.x, 0, &state);

            image_u8x3_cuda_t *d = image_u8x3_create_cuda(w, h);

            for (int i = 0; i < zarray_size_cuda(clusters); i++) {
                zarray_cuda_t *cluster;
                zarray_get_cuda(clusters, i, &cluster);

                uint32_t r, g, b;

                if (1) {
                    const int bias = 50;
                    uint8_t rand1 = (uint8_t) (hiprand_uniform(&state) * (200-bias));
                    uint8_t rand2 = (uint8_t) (hiprand_uniform(&state) * (200-bias));
                    uint8_t rand3 = (uint8_t) (hiprand_uniform(&state) * (200-bias));
                    // uint8_t rand1 = (uint8_t) (10 * (200-bias));
                    // uint8_t rand2 = (uint8_t) (20 * (200-bias));
                    // uint8_t rand3 = (uint8_t) (30 * (200-bias));

                    r = bias + rand1;
                    g = bias + rand2;
                    b = bias + rand3;
                }

                for (int j = 0; j < zarray_size_cuda(cluster); j++) {
                    struct pt *p;
                    zarray_get_volatile_cuda(cluster, j, &p);

                    int x = p->x / 2;
                    int y = p->y / 2;
                    d->buf[y*d->stride + 3*x + 0] = r;
                    d->buf[y*d->stride + 3*x + 1] = g;
                    d->buf[y*d->stride + 3*x + 2] = b;
                }
            }
            
            uint32_t chash = compute_clusters_hash_cuda(clusters);
            uint32_t dbg_hash = compute_image8x3_hash_cuda(d);
            printf("GPU: gradient_cluster hash = 0x%X, dbg image hash = 0x%X\n", chash, dbg_hash);

            *dbg = d;
            // image_u8x3_write_pnm(d, "debug_clusters.pnm");
            // image_u8x3_destroy(d);

        }
    }    

    return NULL;

    if (threadIdx.x == 0) {
        image_u8_destroy_cuda(threshim);
    }

    ////////////////////////////////////////////////////////
    // step 3. process each connected component.
    __shared__ zarray_cuda_t* quads;
    __syncthreads();
    if (threadIdx.x == 0) {
        quads = fit_quads(td, w, h, clusters, im, num_threads);
    } else {
        quads = fit_quads(td, w, h, clusters, im, num_threads);
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        unionfind_destroy_cuda(uf);
    
        for (int i = 0; i < zarray_size_cuda(clusters); i++) {
            zarray_cuda_t *cluster;
            zarray_get_cuda(clusters, i, &cluster);
            zarray_destroy_cuda(cluster);
        }
        zarray_destroy_cuda(clusters);
    }
    return quads;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////
//
// END CUDA version
//
///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////
